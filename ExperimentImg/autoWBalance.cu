#include "hip/hip_runtime.h"
#ifndef  __AUTOWBALANCE_CU_
#define  __AUTOWBALANCE_CU_

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#define datasize 100

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void AutoWBalance(int *In, int *Out, double rgb, int Width, int Height)
{
	//int window[9];
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x <= Width && x >= 0 && y <= Height && y >= 0)
	{

		Out[y* Width + x] = In[y* Width + x]*rgb;
	}
}

extern "C" void AutoWBalance_host(int *pixel,double rgb, int Width, int Height)
{
	int *pixelIn, *pixelOut;
	dim3 dimBlock(32, 32);
	dim3 dimGrid((Width + dimBlock.x - 1) / dimBlock.x, (Height + dimBlock.y -
		1) / dimBlock.y);
	checkCudaErrors(hipMalloc((void**)&pixelIn, sizeof(int) * Width * Height));
	checkCudaErrors(hipMalloc((void**)&pixelOut, sizeof(int) * Width * Height));

	checkCudaErrors(hipMemcpy(pixelIn, pixel, sizeof(int) * Width * Height, hipMemcpyHostToDevice));

	AutoWBalance << <dimGrid, dimBlock >> > (pixelIn, pixelOut,rgb, Width, Height);

	checkCudaErrors(hipMemcpy(pixel, pixelOut, sizeof(int) * Width * Height, hipMemcpyDeviceToHost));


	hipFree(pixelIn);
	hipFree(pixelOut);
}

#endif // ! __AUTOWBALANCE_KERNEL_CU_