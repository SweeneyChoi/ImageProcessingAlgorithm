#include "hip/hip_runtime.h"
#ifndef  __MEDIANFILTER_CU_
#define  __MEDIANFILTER_CU_

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#define datasize 100

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void MedianFilter(int *In, int *Out, int Width, int Height)
{
	int window[9];
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x <= Width && x >= 0 && y <= Height && y >= 0)
	{
		window[0] = (y == 0 || x == 0) ? 0 : In[(y - 1)* Width + x - 1];
		window[1] = (y == 0) ? 0 : In[(y - 1)* Width + x];
		window[2] = (y == 0 || x == Width - 1) ? 0 : In[(y - 1)* Width + x + 1];
		window[3] = (x == 0) ? 0 : In[y* Width + x - 1];
		window[4] = In[y* Width + x];
		window[5] = (x == Width - 1) ? 0 : In[y* Width + x + 1];
		window[6] = (y == Height - 1 || x == 0) ? 0 : In[(y + 1)* Width + x - 1];
		window[7] = (y == Height - 1) ? 0 : In[(y + 1)* Width + x];
		window[8] = (y == Height - 1 || x == Width - 1) ? 0 : In[(y + 1)* Width + x + 1];
		for (unsigned int j = 0; j < 5; j++)
		{
			int min = j;
			for (unsigned int l = j + 1; l < 9; l++)
				if (window[l] < window[min])
					min = l;
			const float temp = window[j];
			window[j] = window[min];
			window[min] = temp;
		}
		Out[y* Width + x] = window[4];
	}
}

extern "C" void MedianFilter_host(int *pixel, int Width, int Height)
{
	int *pixelIn, *pixelOut;
	dim3 dimBlock(32, 32);
	dim3 dimGrid((Width + dimBlock.x - 1) / dimBlock.x, (Height + dimBlock.y -
		1) / dimBlock.y);
	checkCudaErrors(hipMalloc((void**)&pixelIn, sizeof(int) * Width * Height));
	checkCudaErrors(hipMalloc((void**)&pixelOut, sizeof(int) * Width * Height));

	checkCudaErrors(hipMemcpy(pixelIn, pixel, sizeof(int) * Width * Height, hipMemcpyHostToDevice));

	MedianFilter << <dimGrid, dimBlock >> > (pixelIn, pixelOut, Width, Height);

	checkCudaErrors(hipMemcpy(pixel, pixelOut, sizeof(int) * Width * Height, hipMemcpyDeviceToHost));


	hipFree(pixelIn);
	hipFree(pixelOut);
}

#endif // ! __MEDIANFILTER_KERNEL_CU_