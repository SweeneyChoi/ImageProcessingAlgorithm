#include "hip/hip_runtime.h"
#ifndef  __ZOOM_CU_
#define  __ZOOM_CU_

#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <iostream>
#define datasize 100

__device__ inline float BOUND(float val, float min, float max) {
	if (val < min) return min;
	if (val > max) return max;
	return val;
}

__device__ inline float Sinxx(float value) {
	if (value < 0) value = -value;

	if (value < 1.0) {
		float temp = value * value;
		return 0.5 * temp * value - temp + 2.0 / 3.0;
	}
	else if (value < 2.0) {
		value = 2.0 - value;
		value *= value * value;
		return value / 6.0;
	}
	else {
		return 0.0;
	}
}

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void Zoom(int *In, int *Out, int Width, int Height,int nWidth,int nHeight,double counterScale)
{
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x <= nWidth && x >= 0 && y <= nHeight && y >= 0)
	{
		float fx = x * counterScale; //src img��Ӧ���ص�����
		float fy = y * counterScale;
		int rgb;
		if (int(fx) == fx && int(fy) == fy) //����src img�ҵ���Ӧ���ص�
		{
			rgb = In[int(fy)*Width+int(fx)];
		}
		else
		{
			//CubicInterpolation(In, fx, fy, Width, Height,rgb);
			int X, Y;
			X = floor(fx);
			Y = floor(fy);

			int xx[4], yy[4]; //�������굥λ����
			xx[0] = -1;  xx[1] = 0; xx[2] = 1; xx[3] = 2;
			yy[0] = -1;  yy[1] = 0; yy[2] = 1; yy[3] = 2;
			//��֤�Ϸ�
			if ((X - 1) < 0) xx[0] = 0;
			if ((X + 1) > (Width - 1)) xx[2] = 0;
			if ((X + 2) > (Width - 1)) xx[3] = ((xx[2] == 0) ? 0 : 1);

			if ((Y - 1) < 0) yy[0] = 0;
			if ((Y + 1) > (Height - 1)) yy[2] = 0;
			if ((Y + 2) > (Height - 1)) yy[3] = ((yy[2] == 0) ? 0 : 1);

			//�������ص�����ֵ
			int aby[4][4];
			for (int i = 0; i < 4; i++)
			{
				int pbySrcBase = Y + yy[i];
				for (int j = 0; j < 4; j++)
				{
					int pbySrc = pbySrcBase*Width + (X + xx[j]);
					aby[i][j] = In[pbySrc];
				}
			}

			float u, v;
			u = fx - X;
			v = fy - Y;
			//����ֵ��Ȩ��
			float afu[4];
			float afv[4];
			afu[0] = Sinxx(1.0f + u);
			afu[1] = Sinxx(u);
			afu[2] = Sinxx(1.0f - u);
			afu[3] = Sinxx(2.0f - u);

			afv[0] = Sinxx(1.0f + v);
			afv[1] = Sinxx(v);
			afv[2] = Sinxx(1.0f - v);
			afv[3] = Sinxx(2.0f - v);

			//������������м�ֵ
			float af[4] = { 0.0f, 0.0f, 0.0f, 0.0f };

			for (int i = 0; i < 4; i++)
			{
				for (int j = 0; j < 4; j++)
				{
					af[i] += afv[j] * aby[j][i];
				}
			}
			rgb = (int)(BOUND((afu[0] * af[0] + afu[1] * af[1] + afu[2] * af[2] +
				afu[3] * af[3]), 0, 255));
		}
		Out[y* nWidth + x] = rgb;
	}
}

extern "C" void Zoom_host(int *pixel,int *npixel, int Width, int Height,int nWidth,int nHeight,double scaleRate)
{
	int *pixelIn, *pixelOut;
	double counterScale = 1.0 / scaleRate;
	dim3 dimBlock(32, 32);
	dim3 dimGrid((nWidth + dimBlock.x - 1) / dimBlock.x, (nHeight + dimBlock.y -
		1) / dimBlock.y);
	checkCudaErrors(hipMalloc((void**)&pixelIn, sizeof(int) * Width * Height));
	checkCudaErrors(hipMalloc((void**)&pixelOut, sizeof(int) * nWidth * nHeight));

	checkCudaErrors(hipMemcpy(pixelIn, pixel, sizeof(int) * Width * Height, hipMemcpyHostToDevice));

	Zoom << <dimGrid, dimBlock >> > (pixelIn, pixelOut, Width,Height,nWidth, nHeight,counterScale);

	checkCudaErrors(hipMemcpy(npixel, pixelOut, sizeof(int) * nWidth * nHeight, hipMemcpyDeviceToHost));


	hipFree(pixelIn);
	hipFree(pixelOut);
}

#endif // ! __ZOOM_KERNEL_CU_